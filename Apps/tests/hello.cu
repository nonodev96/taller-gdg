#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void cuda_hello()
{
    printf("Hello World from GPU!\n");
}

void print_device_info(int device)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    printf("========================================\n");
    printf(" GPU %d: %s\n", device, prop.name);
    printf("========================================\n");
    printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("  Multi procesadores: %d\n", prop.multiProcessorCount);
    printf("  CUDA Cores (aprox.): %d\n", prop.multiProcessorCount * 128); // Estimación
    printf("  Frecuencia GPU: %.2f MHz\n", prop.clockRate / 1000.0);
    printf("  Memoria global: %.2f GB\n", prop.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
    printf("  Memoria compartida por bloque: %zu KB\n", prop.sharedMemPerBlock / 1024);
    printf("  Tamaño máximo de bloque: %d\n", prop.maxThreadsPerBlock);
    printf("  Número máximo de hilos por multi procesador: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("  Tamaño máximo de grid: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("  Tamaño de warp: %d hilos\n", prop.warpSize);
    printf("  Registros por bloque: %d\n", prop.regsPerBlock);
    printf("  Memoria constante: %zu KB\n", prop.totalConstMem / 1024);
    printf("  Ancho de banda (bus de memoria): %d bits\n", prop.memoryBusWidth);

    printf("  Tensor Cores disponibles: %d\n", prop.asyncEngineCount);       // Disponibilidad de Tensor Cores (si está presente en la GPU)
    printf("  Caché L1 global soportada: %s\n", prop.globalL1CacheSupported ? "Sí" : "No");
    printf("  Caché L1 local soportada: %s\n", prop.localL1CacheSupported ? "Sí" : "No");
    printf("  Tamaño de la caché L2: %.2f KB\n", prop.l2CacheSize / 1024.0); // Tamaño de la caché L2 (si está disponible)
    printf("  GPU parte de una tarjeta multi-GPU: %s\n", prop.isMultiGpuBoard ? "Sí" : "No");
    printf("  GPU multi-GPU ID: %d\n", prop.multiGpuBoardGroupID);
    printf("  Ratio de rendimiento de precisión simple a doble (obsoleto): %d\n", prop.singleToDoublePrecisionPerfRatio);
    printf("  Memoria compartida por bloque (opt-in): %zu KB\n", prop.sharedMemPerBlockOptin / 1024);
    printf("  Memoria compartida reservada por bloque: %zu bytes\n", prop.reservedSharedMemPerBlock);
    printf("  Máximo número de bloques por multi procesador: %d\n", prop.maxBlocksPerMultiProcessor);
    printf("  Se puede usar punteros de host para memoria registrada: %s\n", prop.canUseHostPointerForRegisteredMem ? "Sí" : "No");
    printf("  Acceso concurrente a memoria gestionada: %s\n", prop.concurrentManagedAccess ? "Sí" : "No");
    printf("  Acceso a memoria paginada a través de las tablas de página del host: %s\n", prop.pageableMemoryAccessUsesHostPageTables ? "Sí" : "No");
    printf("  Acceso directo a memoria gestionada desde el host: %s\n", prop.directManagedMemAccessFromHost ? "Sí" : "No");
    printf("  Tamaño máximo de ventana de política de acceso: %d\n", prop.accessPolicyMaxWindowSize);
    printf("  Soporte de operaciones atómicas nativas entre el host y la GPU: %s\n", prop.hostNativeAtomicSupported ? "Sí" : "No");
    printf("  Soporte para preempción de cómputo: %s\n", prop.computePreemptionSupported ? "Sí" : "No");
    printf("  Soporte para CUDA Computed Bindings: %s\n", prop.deviceOverlap ? "Sí" : "No");
    printf("  Soporte para lanzar kernels cooperativos: %s\n", prop.cooperativeLaunch ? "Sí" : "No");
    printf("  Soporte para lanzamiento cooperativo de múltiples dispositivos (obsoleto): %s\n", prop.cooperativeMultiDeviceLaunch ? "Sí" : "No");
    printf("  Soporte para registrar memoria del host: %s\n", prop.hostRegisterSupported ? "Sí" : "No");
    printf("  Soporte para arrays dispersos CUDA: %s\n", prop.sparseCudaArraySupported ? "Sí" : "No");
    printf("  Soporte para registrar memoria como solo lectura: %s\n", prop.hostRegisterReadOnlySupported ? "Sí" : "No");
    printf("  Soporte para interoperabilidad con semáforos de línea de tiempo: %s\n", prop.timelineSemaphoreInteropSupported ? "Sí" : "No");
    printf("  Soporte para pools de memoria: %s\n", prop.memoryPoolsSupported ? "Sí" : "No");
    printf("  Soporte para GPUDirect RDMA: %s\n", prop.gpuDirectRDMASupported ? "Sí" : "No");
    printf("  Soporte para arrays CUDA con mapeo diferido: %s\n", prop.deferredMappingCudaArraySupported ? "Sí" : "No");
    printf("  Soporte para eventos IPC: %s\n", prop.ipcEventSupported ? "Sí" : "No");
    printf("  Opciones de eliminación de RDMA de escritura de GPUDirect: %u\n", prop.gpuDirectRDMAFlushWritesOptions);
    printf("  Orden de escritura RDMA de GPUDirect: %d\n", prop.gpuDirectRDMAWritesOrdering);
    printf("  Tipos de manejadores soportados con IPC basado en pools de memoria: %u\n", prop.memoryPoolSupportedHandleTypes);
    printf("  Lanzamiento de clúster: %s\n", prop.clusterLaunch ? "Sí" : "No");
    printf("  Punteros unificados: %s\n", prop.unifiedFunctionPointers ? "Sí" : "No");
    
    printf("  PCI Bus ID: %d\n", prop.pciBusID);
    printf("  PCI Device ID: %d\n", prop.pciDeviceID);
    printf("  PCI Domain ID: %d\n", prop.pciDomainID);
    
    printf("\n");
}

int cuda_info()
{
    int device_count;
    hipGetDeviceCount(&device_count);

    if (device_count == 0)
    {
        printf("No se encontraron GPUs compatibles con CUDA.\n");
        return 1;
    }    

    int driverVersion = 0;
    int runtimeVersion = 0;
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);

    printf("  Versión del controlador CUDA: %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);
    printf("  Versión del runtime CUDA: %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);

    printf("Se encontraron %d GPU(s) compatibles con CUDA.\n\n", device_count);

    for (int i = 0; i < device_count; i++)
    {
        print_device_info(i);
    }

    return 0;
}

int main()
{
    cuda_hello<<<1, 1>>>();
    cuda_info();
    return 0;
}